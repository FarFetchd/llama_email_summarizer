#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <hip/hip_fp16.h>
#include <atomic>
#include "ggml-cuda.h"

typedef uint16_t ggml_fp16_t;
static_assert(sizeof(__half) == sizeof(ggml_fp16_t), "wrong fp16 size");

#define QK4_0 32
typedef struct {
    float   d;              // delta
    uint8_t qs[QK4_0 / 2];  // nibbles / quants
} block_q4_0;
static_assert(sizeof(block_q4_0) == sizeof(float) + QK4_0 / 2, "wrong q4_0 block size/padding");

#define QK4_1 32
typedef struct {
    float   d;              // delta
    float   m;              // min
    uint8_t qs[QK4_1 / 2];  // nibbles / quants
} block_q4_1;
static_assert(sizeof(block_q4_1) == sizeof(float) * 2 + QK4_1 / 2, "wrong q4_1 block size/padding");

#define QK4_2 16
typedef struct {
    __half  d;              // delta
    uint8_t qs[QK4_2 / 2];  // nibbles / quants
} block_q4_2;
static_assert(sizeof(block_q4_2) == sizeof(ggml_fp16_t) + QK4_2 / 2, "wrong q4_2 block size/padding");

#define QK5_0 32
typedef struct {
    __half d;               // delta
    uint8_t qh[4];          // 5-th bit of quants
    uint8_t qs[QK5_0 / 2];  // nibbles / quants
} block_q5_0;
static_assert(sizeof(block_q5_0) == sizeof(ggml_fp16_t) + sizeof(uint32_t) + QK5_0 / 2, "wrong q5_0 block size/padding");

#define QK5_1 32
typedef struct {
    __half d;               // delta
    __half m;               // min
    uint32_t qh;            // 5-th bit of quants
    uint8_t qs[QK5_1 / 2];  // nibbles / quants
} block_q5_1;
static_assert(sizeof(block_q5_1) == 2 * sizeof(ggml_fp16_t) + sizeof(uint32_t) + QK5_1 / 2, "wrong q5_1 block size/padding");

#define QK8_0 32
typedef struct {
    float   d;              // delta
    int8_t  qs[QK8_0];      // quants
} block_q8_0;
static_assert(sizeof(block_q8_0) == sizeof(float) + QK8_0, "wrong q8_0 block size/padding");

static __global__ void dequantize_block_q4_0(const void * vx, float * y) {
    const block_q4_0 * x = (const block_q4_0 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;

    const uint8_t * pp = x[i].qs;

    for (int l = 0; l < QK4_0; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vi0 = vi & 0xf;
        const int8_t vi1 = vi >> 4;

        const float v0 = (vi0 - 8)*d;
        const float v1 = (vi1 - 8)*d;

        y[i*QK4_0 + l + 0] = v0;
        y[i*QK4_0 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q4_1(const void * vx, float * y) {
    const block_q4_1 * x = (const block_q4_1 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;
    const float m = x[i].m;

    const uint8_t * pp = x[i].qs;

    for (int l = 0; l < QK4_1; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vi0 = vi & 0xf;
        const int8_t vi1 = vi >> 4;

        const float v0 = vi0*d + m;
        const float v1 = vi1*d + m;

        y[i*QK4_1 + l + 0] = v0;
        y[i*QK4_1 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q4_2(const void * vx, float * y) {
    const block_q4_2 * x = (const block_q4_2 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;

    const uint8_t * pp = x[i].qs;

    for (int l = 0; l < QK4_2; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vi0 = vi & 0xf;
        const int8_t vi1 = vi >> 4;

        const float v0 = (vi0 - 8)*d;
        const float v1 = (vi1 - 8)*d;

        y[i*QK4_2 + l + 0] = v0;
        y[i*QK4_2 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q5_0(const void * vx, float * y) {
    const block_q5_0 * x = (const block_q5_0 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;

    const uint8_t * pp = x[i].qs;

    uint32_t qh;
    memcpy(&qh, x[i].qh, sizeof(qh));

    for (int l = 0; l < QK5_0; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vh0 = ((qh & (1 << (l + 0))) >> (l + 0)) << 4;
        const int8_t vh1 = ((qh & (1 << (l + 1))) >> (l + 1)) << 4;

        const int8_t vi0 = ((vi & 0xf) | vh0);
        const int8_t vi1 = ((vi >>  4) | vh1);

        const float v0 = (vi0 - 16)*d;
        const float v1 = (vi1 - 16)*d;

        y[i*QK5_0 + l + 0] = v0;
        y[i*QK5_0 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q5_1(const void * vx, float * y) {
    const block_q5_1 * x = (const block_q5_1 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;
    const float m = x[i].m;

    const uint8_t * pp = x[i].qs;

    const uint32_t qh = x[i].qh;

    for (int l = 0; l < QK5_1; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vh0 = ((qh & (1 << (l + 0))) >> (l + 0)) << 4;
        const int8_t vh1 = ((qh & (1 << (l + 1))) >> (l + 1)) << 4;

        const int8_t vi0 = (vi & 0xf) | vh0;
        const int8_t vi1 = (vi >>  4) | vh1;

        const float v0 = vi0*d + m;
        const float v1 = vi1*d + m;

        y[i*QK5_1 + l + 0] = v0;
        y[i*QK5_1 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q8_0(const void * vx, float * y) {
    const block_q8_0 * x = (const block_q8_0 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;

    const int8_t * pp = x[i].qs;

    for (int l = 0; l < QK8_0; l++) {
        const int8_t vi = pp[l];

        y[i*QK8_0 + l] = vi*d;
    }
}

void dequantize_row_q4_0_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK4_0;
    dequantize_block_q4_0<<<nb, 1, 0, stream>>>(vx, y);
}

void dequantize_row_q4_1_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK4_1;
    dequantize_block_q4_1<<<nb, 1, 0, stream>>>(vx, y);
}

void dequantize_row_q4_2_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK4_2;
    dequantize_block_q4_2<<<nb, 1, 0, stream>>>(vx, y);
}

void dequantize_row_q5_0_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK5_0;
    dequantize_block_q5_0<<<nb, 1, 0, stream>>>(vx, y);
}

void dequantize_row_q5_1_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK5_1;
    dequantize_block_q5_1<<<nb, 1, 0, stream>>>(vx, y);
}

void dequantize_row_q8_0_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK8_0;
    dequantize_block_q8_0<<<nb, 1, 0, stream>>>(vx, y);
}

dequantize_row_q_cuda_t ggml_get_dequantize_row_q_cuda(ggml_type type) {
    switch (type) {
        case GGML_TYPE_Q4_0:
            return dequantize_row_q4_0_cuda;
        case GGML_TYPE_Q4_1:
            return dequantize_row_q4_1_cuda;
        case GGML_TYPE_Q4_2:
            return dequantize_row_q4_2_cuda;
        case GGML_TYPE_Q5_0:
            return dequantize_row_q5_0_cuda;
        case GGML_TYPE_Q5_1:
            return dequantize_row_q5_1_cuda;
        case GGML_TYPE_Q8_0:
            return dequantize_row_q8_0_cuda;
        default:
            return nullptr;
    }
}

// buffer pool for cuda
#define MAX_CUDA_BUFFERS 16

struct scoped_spin_lock {
    std::atomic_flag& lock;
    scoped_spin_lock(std::atomic_flag& lock) : lock(lock) {
        while (lock.test_and_set(std::memory_order_acquire)) {
            ; // spin
        }
    }
    ~scoped_spin_lock() {
        lock.clear(std::memory_order_release);
    }
    scoped_spin_lock(const scoped_spin_lock&) = delete;
    scoped_spin_lock& operator=(const scoped_spin_lock&) = delete;
};

struct cuda_buffer {
    void * ptr = nullptr;
    size_t size = 0;
};

static cuda_buffer g_cuda_buffer_pool[MAX_CUDA_BUFFERS];
static std::atomic_flag g_cuda_pool_lock = ATOMIC_FLAG_INIT;

void * ggml_cuda_pool_malloc(size_t size, size_t * actual_size) {
    scoped_spin_lock lock(g_cuda_pool_lock);

    for (int i = 0; i < MAX_CUDA_BUFFERS; ++i) {
        cuda_buffer& b = g_cuda_buffer_pool[i];
        if (b.size >= size && b.ptr != nullptr) {
            void * ptr = b.ptr;
            *actual_size = b.size;
            b.ptr = nullptr;
            b.size = 0;
            return ptr;
        }
    }
    void * ptr;
    CUDA_CHECK(hipMalloc((void **) &ptr, size));
    *actual_size = size;
    return ptr;
}

void ggml_cuda_pool_free(void * ptr, size_t size) {
    scoped_spin_lock lock(g_cuda_pool_lock);

    for (int i = 0; i < MAX_CUDA_BUFFERS; ++i) {
        cuda_buffer& b = g_cuda_buffer_pool[i];
        if (b.ptr == nullptr) {
            b.ptr = ptr;
            b.size = size;
            return;
        }
    }
    fprintf(stderr, "WARNING: cuda buffer pool full, increase MAX_CUDA_BUFFERS\n");
    CUDA_CHECK(hipFree(ptr));
}

hipblasHandle_t g_cublasH = nullptr;
hipStream_t g_cudaStream = nullptr;
hipStream_t g_cudaStream2 = nullptr;
hipEvent_t g_cudaEvent = nullptr;

void ggml_init_cublas() {
    if (g_cublasH == nullptr) {
        // create cublas handle, bind a stream
        CUBLAS_CHECK(hipblasCreate(&g_cublasH));
        CUDA_CHECK(hipStreamCreateWithFlags(&g_cudaStream, hipStreamNonBlocking));
        CUBLAS_CHECK(hipblasSetStream(g_cublasH, g_cudaStream));

        // create additional stream and event for synchronization
        CUDA_CHECK(hipStreamCreateWithFlags(&g_cudaStream2, hipStreamNonBlocking));
        CUDA_CHECK(hipEventCreateWithFlags(&g_cudaEvent, hipEventDisableTiming));

        // configure logging to stdout
        // CUBLAS_CHECK(cublasLoggerConfigure(1, 1, 0, NULL));
    }
}

hipError_t ggml_cuda_h2d_tensor_2d(void * dst, const struct ggml_tensor * src, uint64_t i3, uint64_t i2, hipStream_t stream) {
    const uint64_t ne0 = src->ne[0];
    const uint64_t ne1 = src->ne[1];
    const uint64_t nb0 = src->nb[0];
    const uint64_t nb1 = src->nb[1];
    const uint64_t nb2 = src->nb[2];
    const uint64_t nb3 = src->nb[3];
    const enum ggml_type type = src->type;
    const size_t ts = ggml_type_size(type);
    const size_t bs = ggml_blck_size(type);

    const void * x = (const void *) ((const char *) src->data + i2*nb2 + i3*nb3);
    if (nb0 == ts && nb1 == ts*ne0/bs) {
        return hipMemcpyAsync(dst, x, ne1*nb1, hipMemcpyHostToDevice, stream);
    } else if (nb0 == ts) {
        return hipMemcpy2DAsync(dst, ts*ne0/bs, x, nb1, ts*ne0/bs, ne1, hipMemcpyHostToDevice, stream);
    } else {
        for (uint64_t i1 = 0; i1 < ne1; i1++) {
            const void * rx = (const void *) ((const char *) x + i1*nb1);
            void * rd = (void *) ((char *) dst + i1*ts*ne0/bs);
            // pretend the row is a matrix with cols=1
            hipError_t r = hipMemcpy2DAsync(rd, ts/bs, rx, nb0, ts/bs, ne0, hipMemcpyHostToDevice, stream);
            if (r != hipSuccess) return r;
        }
        return hipSuccess;
    }
}

void * ggml_cuda_host_malloc(size_t size) {
    void * ptr;
    CUDA_CHECK(hipHostMalloc((void **) &ptr, size));
    return ptr;
}

void ggml_cuda_host_free(void * ptr) {
    CUDA_CHECK(hipHostFree(ptr));
}
